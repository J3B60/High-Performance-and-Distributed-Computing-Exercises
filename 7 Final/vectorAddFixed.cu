#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define N (2048*2048)

#define MIN 0
#define MAX 10

#define THREADS_PER_BLOCK 512

void random_ints(int *a, int n){
    for (int i = 0; i < n; i++) a[i] = rand() % (MAX + 1 - MIN) + MIN;
}

__global__ void add(int *a, int *b, int *c){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N) c[index] = a[index] + b[index];
}

int main(void){
    int *a, *b, *c; // host copies of a, b, c
    int *d_a, *d_b, *d_c; // device copies of a, b, c
    
    int size = N * sizeof(int);
    
    //Alloc space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)%d)c, size);
    
    // Alloc space for host copies of a, b, c and setup input values
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);
    
    random_ints(a, N);
    random_ints(b, N);
    
    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy*d_b, b, size, hipMemcpyHostToDevice);
    
    // Launch add() kernal on GPU with N blocks
    add<<<N, 1>>>(d_a, d_b, d_c);
    
    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    
    // Print Results
    for(int i=0; i<N; i++) printf("%2d+%2d=%2d\n", a[i], b[i], c[i]);
    
    // Cleanup
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    
    return 0;
}