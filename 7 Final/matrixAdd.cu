#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define N 4
#define MIN 1
#define MAX 9

void random_ints(int a[][N]){
    // Remeber this nested for loop is single line so it just ends with ';', it doesn't need '{' and '}'
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
            a[i][j] = rand() % (MAX + 1 - MIN) + MIN;
}

void print(int a[][N]){
    for(int i=0; i<N; i++){
        for(int j=0; j<N; j++) 
            printf("%2d", a[i][j]);
        printf("\n");
    }
}

__global__ void matAdd(int a[][N], int b[][N], int c[][N]){
    // insert your code here
}

int main(void) {
    int a[N][N], b[N][N], c[N][N]; // host copies of a, b, c
    int (*d_a)[N], (*d_b)[N], (*d_c)[N];  // device copies of a, b, c
    
    int size = (N * N) * sizeof(int);
    
    // Alloc space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
    
    random_ints(a);
    random_ints(b);
    
    // Copy inputs to device
    hipMemcpy(d_a, a, size, cudaMemHostToDevice);
    hipMemcpy(d_b, b, size, cudaMemHostToDevice);
    
    int numBlocks = 1;
    dim3 threadsPerBlock(N, N);
    
    // Launch add() kernal on GPU with N blocks
    matAdd<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c);
    
    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    
    // Print Result
    print(a); printf("+\n"); print(b); printf("=\n"); print(c);
    
    // Cleanup
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    
    return 0;
}